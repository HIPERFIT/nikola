#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include <inttypes.h>

#define CS_X(i) (lowx + (double) (i) * (highx - lowx) / (double) viewx)
#define CS_Y(j) (lowy + (double) (j) * (highy - lowy) / (double) viewy)

__device__ unsigned int blockCounter;   // global counter, initialized to zero
                                        // before kernel launch

extern "C" static __global__ void stepN(double lowx, double lowy, double highx, double highy,
                                        int32_t viewx, int32_t viewy,
                                        double* cs_x, double* cs_y,
                                        int32_t cs_dimx, int32_t cs_dimy,
                                        double* zs_x, double* zs_y, int32_t* zs_i,
                                        int32_t zs_dimx, int32_t zs_dimy,
                                        int32_t depth,
                                        const int gridWidth, const int numBlocks)
{
    __shared__ unsigned int blockIndex;
    __shared__ unsigned int blockX, blockY;

    while(1) {
        if ((threadIdx.x==0) && (threadIdx.y==0)) {
            // get block to process
            blockIndex = atomicAdd(&blockCounter, 1);
            blockX = blockIndex % gridWidth;            // note: this is slow, but only called once per block here
            blockY = blockIndex / gridWidth;
        }
        __syncthreads();

        if (blockIndex >= numBlocks) break;  // finish

        // process this block
        const int i = blockDim.x * blockX + threadIdx.x;
        const int j = blockDim.y * blockY + threadIdx.y;

        int32_t idx;
        double x_temp;
        double y_temp;
        int32_t i_temp;
        double cs_x = CS_X(i);
        double cs_y = CS_Y(j);

        idx  = j * zs_dimx + i;
        x_temp = cs_x;
        y_temp = cs_y;
        i_temp = 0;
        if (depth > 0) {
            for (int k = 0; k < depth; ++k) {
                int32_t idx3;
                double x_new;
                double y_new;
                double xx = x_temp * x_temp;
                double xy = x_temp * y_temp;
                double yy = y_temp * y_temp;

                idx3 = j * cs_dimx + i;
                x_new = cs_x + xx - yy;
                y_new = cs_y + xy + xy;

                if (x_new * x_new + y_new * y_new > 4.0)
                    break;

                x_temp = x_new;
                y_temp = y_new;
                i_temp = 1 + i_temp;
            }
        }
        zs_x[idx] = x_temp;
        zs_y[idx] = y_temp;
        zs_i[idx] = i_temp;
    }
}

#define DIM_X   2
#define DIM_Y   2
#define BLOCK_X 16
#define BLOCK_Y 16

hipError_t mandelbrot_cuda_4(double lowx, double lowy, double highx, double highy,
                              int32_t viewx, int32_t viewy, int32_t depth,
                              double* cs_x, double* cs_y,
                              int32_t cs_dimx, int32_t cs_dimy,
                              double* zs_x, double* zs_y,
                              int32_t* zs_i, int32_t zs_dimx,
                              int32_t zs_dimy)
{
    {
        dim3 gridDims;
        dim3 blockDims;

        blockDims.x = DIM_X;
        gridDims.x = BLOCK_X;
        blockDims.y = DIM_Y;
        gridDims.y = BLOCK_Y;
        stepN<<<blockDims, gridDims>>>(lowx, lowy, highx, highy,
                                       viewx, viewy,
                                       cs_x, cs_y,
                                       cs_dimx, cs_dimy,
                                       zs_x, zs_y, zs_i,
                                       zs_dimx, zs_dimy,
                                       depth,
                                       viewx, (viewx*viewy)/(BLOCK_X*BLOCK_Y));
    }

    return hipGetLastError();
}
